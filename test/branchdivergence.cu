
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>

__global__
void branchtest(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n and threadIdx.x != 0)
    y[i] = a * x[i] + y[i];
}

using namespace std;
using namespace chrono;

#define CU_CHK(ERRORCODE) \
{hipError_t error = ERRORCODE; \
  if (error != 0) \
  { cerr << hipGetErrorName(error) << ": " << hipGetErrorString(error) << \
    " at " << __FILE__ << ":" << __LINE__ << "\n";}}

int main(int argc, char **argv) {
  int N = 1 * (1 << 20);
  float *x, *y, *res, *d_x, *d_y;
  x = (float *) malloc(N * sizeof(float));
  y = (float *) malloc(N * sizeof(float));
  res = (float *) malloc(N * sizeof(float));

  CU_CHK(hipMalloc(&d_x, N * sizeof(float)));
  CU_CHK(hipMalloc(&d_y, N * sizeof(float)));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f + float(i);
  }

  cout << "CTAs: " << (N + 511) / 512 << "\n";

  auto t0 = high_resolution_clock::now();
  CU_CHK(hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));
  CU_CHK(hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice));
  auto t1 = high_resolution_clock::now();

  branchtest <<< (N + 511) / 512, 512 >>> (N, 2.0f, d_x, d_y);
  CU_CHK(hipGetLastError());
  CU_CHK(hipDeviceSynchronize());
  auto t2 = high_resolution_clock::now();

  CU_CHK(hipMemcpy(res, d_y, N * sizeof(float), hipMemcpyDeviceToHost));
  auto t3 = high_resolution_clock::now();

  if (hipGetLastError() != 0)
    return -1;

  for (int i = 0; i < N; i++) {
    float y_host = 2.0f * x[i] + y[i];
    float diff = y_host - res[i];
    if (diff > 1.0f)
      cout << "Error at y[" << i << "]: " << y_host << " vs. " << res[i] << "\n";
  }

  cout << "Duaration memcpy to device: " << duration_cast<microseconds>(t1 - t0).count() << endl;
  cout << "Duaration kernel: " << duration_cast<microseconds>(t2 - t1).count() << endl;
  cout << "Duaration memcpy to host: " << duration_cast<microseconds>(t3 - t1).count() << endl;

  // write to file
  if (argc > 1) {
    ofstream file(argv[1], std::ios::binary);
    file.write((char *) y, N * sizeof(float));
  }

  return 0;
}
